#include "hip/hip_runtime.h"
#include "TensorCoreCudaHead.cuh"

__global__ void AddArrayKernel(float* Output, float* InputFirst, float* InputSecond,size_t Size) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < Size) Output[Index] = InputFirst[Index] + InputSecond[Index];
}

__global__ void FillArrayKernel(float* Input, float Scalar,size_t Size) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < Size) Input[Index] = Scalar;
}

__global__ void DotArrayKernel(float* Output, size_t OutSize, size_t InSize) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < InSize && Index + InSize < OutSize)Output[Index] += Output[Index + InSize];
  __syncthreads();
}

__global__ void AddScalarKernel(float* Output,float* Input, float Scalar,size_t Size) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < Size) Output[Index] = Input[Index] + Scalar;
}

__global__ void MulScalarKernel(float* Output,float* Input, float Scalar,size_t Size) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < Size) Output[Index] = Input[Index] * Scalar;
}

__global__ void AddKernel(float* Output, float* HighDimInput, size_t HighDimSize, float* LowDimInput, size_t LowDimSize) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < HighDimSize)Output[Index] = HighDimInput[Index] + LowDimInput[Index%LowDimSize];
}

__global__ void EleMulKernel(float* Output, float* HighDimInput, size_t HighDimSize, float* LowDimInput, size_t LowDimSize) 
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < HighDimSize)Output[Index] = HighDimInput[Index] * LowDimInput[Index%LowDimSize];
}

__global__ void TKernel(float* Output, float* Input, size_t *MatrixShape, size_t ShapeCount)
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if(Index < ShapeCount)
  {
    size_t MatrixShapeCount = MatrixShape[0]*MatrixShape[1];
    size_t InputMatIndex = Index%MatrixShapeCount;
    size_t BaseCount = Index - InputMatIndex;
    size_t InputMatIndexFirst = InputMatIndex/MatrixShape[1];
    size_t InputMatIndexSecond = InputMatIndex%MatrixShape[1];
    Output[BaseCount + InputMatIndexSecond*MatrixShape[0] + InputMatIndexFirst] = Input[Index];
  }
}

__global__ void MatmulKernel
(
  float* Output, 
  size_t *OutputBatchShape, 
  size_t *OutputMatrixShape,
  float* InputFirst, 
  size_t *InputFirstBatchShape, 
  size_t *InputFirstMatrixShape,
  float* InputSecond, 
  size_t *InputSecondBatchShape, 
  size_t *InputSecondMatrixShape,
  size_t BatchShapeLen,
  size_t OutputShapeCount
)
{
  size_t Index = blockIdx.x * blockDim.x + threadIdx.x;
  if (Index < OutputShapeCount)
  {
    size_t OutputBatchIndex[8];
    size_t OutputMatrixShapeCount = OutputMatrixShape[0]*OutputMatrixShape[1];
    size_t OutSizeTMP = Index/OutputMatrixShapeCount;
    bool MatZero = OutSizeTMP;
    for(int a=BatchShapeLen-1;a>=0;a--)
    {
      if(!MatZero)OutputBatchIndex[a] = 0;
      else
      {
        OutputBatchIndex[a] = OutSizeTMP%OutputBatchShape[a];
        OutSizeTMP /= OutputBatchShape[a];
      }
    }
    size_t InputFirstBatchIndex[8];
    for(int a=BatchShapeLen-1;a>=0;a--)
    {
      if(OutputBatchIndex[a] < InputFirstBatchShape[a])InputFirstBatchIndex[a] = OutputBatchIndex[a];
      else InputFirstBatchIndex[a] = 0;
    }
    size_t InputFirstMatrixShapeCount = InputFirstMatrixShape[0]*InputFirstMatrixShape[1];
    size_t InputSecondBatchIndex[8];
    for(int a=BatchShapeLen-1;a>=0;a--)
    {
      if(OutputBatchIndex[a] < InputSecondBatchShape[a])InputSecondBatchIndex[a] = OutputBatchIndex[a];
      else InputSecondBatchIndex[a] = 0;
    }
    size_t InputSecondMatrixShapeCount = InputSecondMatrixShape[0]*InputSecondMatrixShape[1];
    size_t InputFirstBase = 0;
    size_t InFirstTMP = InputFirstMatrixShapeCount;
    for(int a=BatchShapeLen-1;a>=0;a--)
    {
      InputFirstBase += InFirstTMP*InputFirstBatchIndex[a];
      InFirstTMP*=InputFirstBatchShape[a];
    }
    size_t InputSecondBase = 0;
    size_t InSecondTMP = InputSecondMatrixShapeCount;
    for(int a=BatchShapeLen-1;a>=0;a--)
    {
      InputSecondBase += InSecondTMP*InputSecondBatchIndex[a];
      InSecondTMP*=InputSecondBatchShape[a];
    }
    size_t OutputMatrixIndex = Index%OutputMatrixShapeCount;
    size_t MatIndex[2] = {OutputMatrixIndex/OutputMatrixShape[1], OutputMatrixIndex%OutputMatrixShape[1]};
    Output[Index] = 0;
    for(int a=0;a<InputFirstMatrixShape[1];a++)
    {
      Output[Index] += InputFirst[InputFirstBase + MatIndex[0]*InputFirstMatrixShape[1] + a]*InputSecond[InputSecondBase + a*InputSecondMatrixShape[1] + MatIndex[1]];
    }
  }
}

void TInCPP(float* Output, float* Input, size_t *MatrixShape, size_t ShapeCount)
{
  size_t *MatrixShapeCuda;
  hipMalloc((void**)&MatrixShapeCuda, 2*sizeof(size_t));
  hipMemcpy(MatrixShapeCuda,MatrixShape,sizeof(size_t)*2,hipMemcpyHostToDevice);
  CudaPair CudaPairInput = GetCudaPair(ShapeCount);
  TKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Output, Input, MatrixShapeCuda, ShapeCount);
  hipFree(MatrixShapeCuda);
}

void MatmulInCPP
(
  float* Output, 
  size_t OutputBatchShape[8], 
  size_t OutputMatrixShape[2],
  float* InputFirst, 
  size_t InputFirstBatchShape[8], 
  size_t InputFirstMatrixShape[2],
  float* InputSecond, 
  size_t InputSecondBatchShape[8], 
  size_t InputSecondMatrixShape[2],
  size_t BatchShapeLen,
  size_t OutputShapeCount,
  size_t DeviceNum
)
{
  hipSetDevice(DeviceNum);
  size_t *OutputBatchShapeCuda;
  hipMalloc((void**)&OutputBatchShapeCuda, 8*sizeof(size_t));
  hipMemcpy(OutputBatchShapeCuda,OutputBatchShape,sizeof(size_t)*8,hipMemcpyHostToDevice);
  size_t *OutputMatrixShapeCuda;
  hipMalloc((void**)&OutputMatrixShapeCuda, 2*sizeof(size_t));
  hipMemcpy(OutputMatrixShapeCuda,OutputMatrixShape,sizeof(size_t)*2,hipMemcpyHostToDevice);
  size_t *InputFirstBatchShapeCuda;
  hipMalloc((void**)&InputFirstBatchShapeCuda, 8*sizeof(size_t));
  hipMemcpy(InputFirstBatchShapeCuda,InputFirstBatchShape,sizeof(size_t)*8,hipMemcpyHostToDevice);
  size_t *InputFirstMatrixShapeCuda;
  hipMalloc((void**)&InputFirstMatrixShapeCuda, 2*sizeof(size_t));
  hipMemcpy(InputFirstMatrixShapeCuda,InputFirstMatrixShape,sizeof(size_t)*2,hipMemcpyHostToDevice);
  size_t *InputSecondBatchShapeCuda;
  hipMalloc((void**)&InputSecondBatchShapeCuda, 8*sizeof(size_t));
  hipMemcpy(InputSecondBatchShapeCuda,InputSecondBatchShape,sizeof(size_t)*8,hipMemcpyHostToDevice);
  size_t *InputSecondMatrixShapeCuda;
  hipMalloc((void**)&InputSecondMatrixShapeCuda, 2*sizeof(size_t));
  hipMemcpy(InputSecondMatrixShapeCuda,InputSecondMatrixShape,sizeof(size_t)*2,hipMemcpyHostToDevice);
  CudaPair CudaPairInput = GetCudaPair(OutputShapeCount);
  MatmulKernel<<<CudaPairInput.block, CudaPairInput.grid>>>
  (
    Output, 
    OutputBatchShapeCuda, 
    OutputMatrixShapeCuda, 
    InputFirst,
    InputFirstBatchShapeCuda, 
    InputFirstMatrixShapeCuda,
    InputSecond, 
    InputSecondBatchShapeCuda,
    InputSecondMatrixShapeCuda,
    BatchShapeLen,
    OutputShapeCount
  );
  hipFree(OutputBatchShapeCuda);
  hipFree(OutputMatrixShapeCuda);
  hipFree(InputFirstBatchShapeCuda);
  hipFree(InputFirstMatrixShapeCuda);
  hipFree(InputSecondBatchShapeCuda);
  hipFree(InputSecondMatrixShapeCuda);
}

void FillArrayInCPP(float* Input, float Scalar,size_t Size)
{
  CudaPair CudaPairInput = GetCudaPair(Size);
  FillArrayKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Input, Scalar, Size);
}

void AddScalarInCPP(float* Output,float* Input, float Scalar,size_t Size) 
{
  CudaPair CudaPairInput = GetCudaPair(Size);
  AddScalarKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Output,Input, Scalar, Size);
}

void MulScalarInCPP(float* Output,float* Input, float Scalar,size_t Size) 
{
  CudaPair CudaPairInput = GetCudaPair(Size);
  MulScalarKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Output,Input, Scalar, Size);
}

void AddArrayInCPP(float* Output, float* InputFirst, float* InputSecond, size_t Size) 
{
  CudaPair CudaPairInput = GetCudaPair(Size);
  AddArrayKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Output, InputFirst, InputSecond, Size);
}

void DotArrayInCPP(float* Output, float* InputFirst, float* InputSecond, size_t Size) 
{
  CudaPair CudaPairInput = GetCudaPair(Size);
  float *OutTMP;
  hipMalloc((void**)&OutTMP, Size*sizeof(float));
  EleMulKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(OutTMP, InputFirst, Size, InputSecond, Size);
  size_t SizeTMP = Size;
  while(SizeTMP > 1)
  {
    CudaPairInput = GetCudaPair(SizeTMP);
    DotArrayKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(OutTMP, SizeTMP, (SizeTMP + 1)/2);
    SizeTMP = (SizeTMP + 1)/2;
  }
  hipMemcpy(Output,OutTMP,sizeof(float),hipMemcpyDeviceToDevice);
  hipFree(OutTMP);
}

void AddInCPP(float* Output, float* HighDimInput, size_t HighDimSize, float* LowDimInput, size_t LowDimSize) 
{
  CudaPair CudaPairInput = GetCudaPair(HighDimSize);
  AddKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Output, HighDimInput, HighDimSize, LowDimInput, LowDimSize);
}

void EleMulInCPP(float* Output, float* HighDimInput, size_t HighDimSize, float* LowDimInput, size_t LowDimSize) 
{
  CudaPair CudaPairInput = GetCudaPair(HighDimSize);
  EleMulKernel<<<CudaPairInput.block, CudaPairInput.grid>>>(Output, HighDimInput, HighDimSize, LowDimInput, LowDimSize);
}

void DataToCPU(float* CPUPointer, float* GPUPointer, size_t Size){hipMemcpy(CPUPointer,GPUPointer,sizeof(float)*Size,hipMemcpyDeviceToHost);}
void DataToGPU(float* CPUPointer, float* GPUPointer, size_t Size){hipMemcpy(GPUPointer,CPUPointer,sizeof(float)*Size,hipMemcpyHostToDevice);}
void cudaFreeInCPP(float* Input){hipFree(Input);}
void cudaMallocInCPP(float** Input, size_t Size, size_t DeviceNum)
{
  hipSetDevice(DeviceNum);
  hipMalloc(Input, Size*sizeof(float));
}




